// Copyright (C) 2024 ByteDance and/or its affiliates
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//          http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <string>
#include <sstream>
#include "mma_any/aq_bmma_library.h"
#include "mma_any/aq_bmma_op.h"
#include "test/test_mma/test_mma.h"

void test_mma_w2a16(int x_bits, int w_bits, int *d_x, int *d_w, int *d_x_pack, int *d_w_pack, int m,
                    int n, int k, int *d_out, int *h_out, int *h_ref_out, int warmup, int repeat,
                    bool quant_sign, hipStream_t stream)
{
#ifdef W2A16
    std::string config_str;
    std::stringstream s;
    s << x_bits << " " << w_bits << " " << m << " " << n << " " << k << " ";
    if (quant_sign) {
        s << "sign ";
    } else {
        s << "unsigned ";
    }
    config_str = s.str();
    float exec_dur = 0;
    float pack_dur = 0;
    float true_gflop_count = (float)m / 1e9 * n * k * 2 * x_bits * w_bits;
    float gflop_count = (float)m / 1e9 * n * k * 2;
    float max_gflop = 0;
    std::stringstream best_config;

    if (quant_sign) {
        ////// W2A8 int
        // cta<1,32,256> warp<8,32,128> mma<8,8,128>   WARPS[1x2]
        TEST(16, 2, true, 1, 32, 256, 8, 32, 128, 8, 8, 128, 2);

    } else {
    }

    printf("The best kernel config is %s with %f TOPS\n", best_config.str().c_str(), max_gflop);
#else
    printf("unsupport w%da%d\n", w_bits, x_bits);
#endif
}